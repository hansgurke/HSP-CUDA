#include "hip/hip_runtime.h"

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
//#include "Quaternion.h"
#include "Constants.h"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


/*START CLASS DEFINITIONS*/
/**/
/**/

class Quaternion
{

	public:

		__device__ Quaternion(float ar, float ai, float aj, float ak)
			{
				real=ar;
				i=ai;
				j=aj;
				k=ak;
			};

		__device__ Quaternion operator+(Quaternion b)
			{
				return Quaternion(real + b.real, i + b.i, j + b.j, k + b.k);
			};

		__device__ Quaternion operator*(Quaternion b)
			{
				float cr = real*b.real - i*b.i - j*b.j - k*b.k;
				float ci = real*b.i + i*b.real + j*b.k - k*b.j;
				float cj = real*b.j - i*b.k + j*b.real + k*b.i;
				float ck = real*b.k + i*b.j - j*b.i + k*b.real;
				return Quaternion(cr, ci, cj, ck);
			};

		__device__ Quaternion operator=(Quaternion b)
			{
				real=b.real;
				i=b.i;
				j=b.j;
				k=b.k;
				return *this;
			};

		__device__ Quaternion operator+=(Quaternion b)
			{
				real+=b.real;
				i+=b.i;
				j+=b.j;
				k+=b.k;
				return *this;
			};

		__device__ Quaternion operator*=(Quaternion b)
			{
				float cr = real*b.real - i*b.i - j*b.j - k*b.k;
				float ci = real*b.i + i*b.real + j*b.k - k*b.j;
				float cj = real*b.j - i*b.k + j*b.real + k*b.i;
				float ck = real*b.k + i*b.j - j*b.i + k*b.real;
				real=cr;
				i=ci;
				j=cj;
				k=ck;
				return *this;
			};

		__device__  float abs(void)
			{
				return sqrt(pow(real,2)+pow(i,2)+pow(j,2)+pow(k,2));
			};
		__device__ ~Quaternion()
			{

			};


	private:

		float real;
		float i;
		float j;
		float k;
};



/*END CLASS DEFINITIONS*/
/**/
/**/



/*START GLOBAL DEFINITIONS*/
/**/
/**/

__host__ __device__ float getCoordinateValue(int Index)
{
	return ((float)(Index*4)/(float)(DIMENSION-1))-2;
}

__host__ __device__ int getXIndexFromArrayIndex(int Index)
{
	return (int)Index/(DIMENSION*DIMENSION);
}

__host__ __device__ int getYIndexFromArrayIndex(int Index)
{
	return (int)(Index-DIMENSION)/DIMENSION;
}

__host__ __device__ int getZIndexFromArrayIndex(int Index)
{
	return (int)(Index-DIMENSION-DIMENSION)/DIMENSION;
}


/**/
/**/
/*END GLOBAL DEFINITIONS*/



/*START KERNEL DEFINITION*/
/**/
/**/

__global__ void calc_JuliaSet_quat_3D_Part(unsigned char* A, float k_Index, float C_real, float C_i, float C_j, float C_k)
{
	unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned long d = DIMENSION;
	unsigned long DIM = d*d*d;
	if(tid+(blockDim.x*gridDim.x)<DIM)
	{
		unsigned int number_of_works=DIM/(blockDim.x*gridDim.x);

		for(int i=0; i<=number_of_works; i++)
		{
			Quaternion Z = Quaternion(getXIndexFromArrayIndex(tid+(i*blockDim.x*gridDim.x)),getXIndexFromArrayIndex(tid+(i*blockDim.x*gridDim.x)), getXIndexFromArrayIndex(tid+(i*blockDim.x*gridDim.x)), k_Index);
			Quaternion C = Quaternion(C_real, C_i, C_j, C_k);
			//function to calculate MandelbrotSet  z(1) = z(0)² + c
			int k=0;
			while(k<MAX_ITERATIONS && Z.abs()<2)
			{
				Z=Z*Z+C;
				k++;
			}
			A[tid+(i*blockDim.x*gridDim.x)]=k/(MAX_ITERATIONS/256);
		}
	}
}

/**/
/**/
/*END KERNEL DEFINITION*/






void start_Calculation()
{
	//Quaternion C = Quaternion(0.1, 0.1, 0.1, 0.1);
	int devices = 0;
	unsigned char* host_arrays[MAX_DEVICES_POSSIBLE];
	unsigned char* device_arrays[MAX_DEVICES_POSSIBLE];
	size_t size = DIMENSION * DIMENSION * DIMENSION * sizeof(unsigned char);
	hipError_t error;
	//float k_index = 0;

	/*Cuda Pre-Condition-Checking*/
	/**/

	//Look how many Devices are present
	error = hipGetDeviceCount(&devices);
	if (error != hipSuccess)
	{
	    printf("hipGetDeviceCount returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	    exit(EXIT_FAILURE);
	}
	if(devices>MAX_DEVICES_POSSIBLE)
	{
		printf("too much devices found! increment MAX_DEVICES_POSSIBLE");
		exit(EXIT_FAILURE);
	}
	printf("DeviceCount: %d\n", devices);

	//create Array for each Device
	for(int i=0; i<devices; i++)
	{
		host_arrays[i] = (unsigned char *)malloc(size);

		//set context to specific device
		error = hipSetDevice(i);
		if (error != hipSuccess)
		{
		    printf("cudaSetDeviceCount returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    exit(EXIT_FAILURE);
		}

		//allocate memory on device
		error = hipMalloc((void **) &device_arrays[i], size);
		if (error != hipSuccess)
		{
		    printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    exit(EXIT_FAILURE);
		}
	}

	int pos=0;
	while(pos<DIMENSION)
	{
		//starting Kernels
		for(int i=0; i<devices; i++)
		{
			//set context to specific device
			error = hipSetDevice(i);
			if (error != hipSuccess)
			{
			    printf("cudaSetDeviceCount returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			    exit(EXIT_FAILURE);
			}
			//printf("%f\n", getCoordinateValue(pos));
			calc_JuliaSet_quat_3D_Part<<<MAX_BLOCKS_PER_GRID, MAX_THREADS_PER_BLOCK>>>(device_arrays[i], getCoordinateValue(pos), 0.1, 0.1, 0.1, 0.1);
			pos++;
		}
		/*
		if(pos%20==0)
		{
			printf("started %d Kernels\n position is now %d\n", devices, pos);
		}
		*/
		//reading back results
		for(int i=0; i<devices; i++)
		{
			//set context to specific device
			error = hipSetDevice(i);
			if (error != hipSuccess)
			{
			    printf("cudaSetDeviceCount returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			    exit(EXIT_FAILURE);
			}
			//printf("hipMemcpy");
			hipMemcpy(host_arrays[i], device_arrays[i], size, hipMemcpyDeviceToHost);
		}
	}

	//free all host and device array memory
	for(int i=0; i<devices; i++)
	{
		//set context to specific device
		error = hipSetDevice(i);
		if (error != hipSuccess)
		{
		    printf("cudaSetDeviceCount returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		    exit(EXIT_FAILURE);
		}
		hipFree(device_arrays[i]);
		free(host_arrays[i]);
	}


}



int main(int argc, char* argv[])
{
	clock_t prgstart, prgende;
	printf("start with %d DIMs and %d BLOCKS\n", DIMENSION, MAX_BLOCKS_PER_GRID);
	prgstart=clock();
	start_Calculation();
	prgende=clock();//CPU-Zeit am Ende des Programmes
	printf("Laufzeit %.2f Sekunden\n",(float)(prgende-prgstart) / CLOCKS_PER_SEC);
	printf("stop\n");
	return 0;
}
